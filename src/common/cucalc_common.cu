
#include <hip/hip_runtime.h>
#include <stdio.h>

// #define gpuErrchk(val) cudaErrorCheck(val, __FILE__, __LINE__, true)
void cudaErrorCheck(hipError_t err, const char* message, bool abort) {
  if (err != hipSuccess) {
    printf("%s:%s\n%s\n", hipGetErrorName(err), hipGetErrorString(err), message);
    if (abort) exit(-1);
  }
}